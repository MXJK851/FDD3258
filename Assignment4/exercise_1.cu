
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 256
#define TPB 256

__global__ void cuda_hello()
{
  const int myID = blockIdx.x*blockDim.x+threadIdx.x;
  printf("Hello World! My threadId is %d\n",myID);
}

int main()
{ 
  // Allocate device memory to store the output array
  // cudaMalloc(&d_out, N*sizeof(float));
  
  // Launch kernel to compute and store distance values
  cuda_hello<<<N/TPB, TPB>>>();
  
  hipDeviceSynchronize();
  
  // cudaFree(d_out); // Free the memory
  
  return 0;
}
