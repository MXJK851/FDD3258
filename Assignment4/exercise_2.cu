
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#define ARRAY_SIZE 97
#define TPB 32

__host__ float cpu_saxpy(int i, float a, float *X, float *Y)
{
  return (a*X[i]+Y[i]);
}

__device__ float gpu_saxpy(int i, float a, float *X, float *Y)
{
  return (a*X[i]+Y[i]);
}

__global__ void ThreadId(float *y_out, int n, float a, float *X, float *Y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  if (i<n)
  {
    y_out[i]=gpu_saxpy(i,a,X,Y);
  } 

}

int main()
{
  float a=2.0,X[ARRAY_SIZE],*x=0,Y[ARRAY_SIZE],*y=0,Y_out[ARRAY_SIZE],*y_out=0;
 
  int j;
  
  for (j=0;j<ARRAY_SIZE;j++)
  {
    X[j]=j;
    Y[j]=j;
    Y_out[j]=cpu_saxpy(j,a,X,Y);
    // printf("%f\n",Y_out[j]);
  }

  hipMalloc(&y_out, ARRAY_SIZE*sizeof(float));
  hipMalloc(&x, ARRAY_SIZE*sizeof(float));
  hipMalloc(&y, ARRAY_SIZE*sizeof(float));

  hipMemcpy(x, X, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(y, Y, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
  
  int num_block;
  num_block= ARRAY_SIZE/TPB;
  while (ARRAY_SIZE>num_block*TPB)
  {
    num_block++;
  }

  //num_block=ceil(float(ARRAY_SIZE/TPB));
  //printf("%d\n",num_block);
  //ThreadId<<<ceil(ARRAY_SIZE/TPB), TPB>>>(y_out, ARRAY_SIZE, a, x, y);
  ThreadId<<<num_block,TPB>>>(y_out, ARRAY_SIZE, a, x, y);

  hipDeviceSynchronize();  
 
  //float *y=(float*)malloc(ARRAY_SIZE*sizeof(float));
  float z[ARRAY_SIZE];
  
  for (j=0;j<ARRAY_SIZE;j++)
  {
    hipMemcpy(z, y_out, ARRAY_SIZE*sizeof(float), hipMemcpyDeviceToHost);
  }

  hipFree(y_out);
  hipFree(x);
  hipFree(y);
 
  for (j=0;j<ARRAY_SIZE;j++)
  { 
    printf("CPU:%4.1f  GPU:%4.1f  Compare:%4.1f\n",Y_out[j],z[j],Y_out[j]-z[j]);
  }

  return 0;
}
