
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#define ARRAY_SIZE 100000
#define TPB 32

__host__ float cpu_saxpy(int i, float a, float *X, float *Y)
{
  return (a*X[i]+Y[i]);
}

__device__ float gpu_saxpy(int i, float a, float *X, float *Y)
{
  return (a*X[i]+Y[i]);
}

__global__ void ThreadId(float *y_out, int n, float a, float *X, float *Y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  if (i<n)
  {
    y_out[i]=gpu_saxpy(i,a,X,Y);
  } 

}

int main()
{
  float a=2.0,X[ARRAY_SIZE],*x=0,Y[ARRAY_SIZE],*y=0,Y_out[ARRAY_SIZE],*y_out=0;
 
  int j;

  struct timeval time1, time2, time3, time4;
  
  for (j=0;j<ARRAY_SIZE;j++)
  {
    X[j]=j;
    Y[j]=j;
  }
  
  gettimeofday(&time1, NULL);

  for (j=0;j<ARRAY_SIZE;j++)
  {
    Y_out[j]=cpu_saxpy(j,a,X,Y);
  }

  gettimeofday(&time2, NULL);

  hipMalloc(&y_out, ARRAY_SIZE*sizeof(float));
  hipMalloc(&x, ARRAY_SIZE*sizeof(float));
  hipMalloc(&y, ARRAY_SIZE*sizeof(float));

  hipMemcpy(x, X, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(y, Y, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
  
  int num_block;
  num_block= ARRAY_SIZE/TPB;
  while (ARRAY_SIZE>num_block*TPB)
  {
    num_block++;
  }

  gettimeofday(&time3, NULL);

  ThreadId<<<num_block,TPB>>>(y_out, ARRAY_SIZE, a, x, y);

  gettimeofday(&time4, NULL);

  hipDeviceSynchronize();  
 
  float z[ARRAY_SIZE];
  
  for (j=0;j<ARRAY_SIZE;j++)
  {
    hipMemcpy(z, y_out, ARRAY_SIZE*sizeof(float), hipMemcpyDeviceToHost);
  }

  hipFree(y_out);
  hipFree(x);
  hipFree(y);
 
  for (j=0;j<ARRAY_SIZE;j++)
  { 
    printf("CPU:%4.1f  GPU:%4.1f  Compare:%4.1f\n",Y_out[j],z[j],Y_out[j]-z[j]);
  }
  
  printf("CPU Execution: %ldms  GPU Execution: %ldms\n",time2.tv_usec-time1.tv_usec,time4.tv_usec-time3.tv_usec);

  return 0;
}
